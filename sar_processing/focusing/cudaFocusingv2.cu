#include "hip/hip_runtime.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <iostream>

#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line)
{
    if (result)
    {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        mexErrMsgTxt("CUDA Error,exit.");
    }
}

__device__ float gaussActivFunc(float x, float sigma)
{
    float out = expf(-0.5 * (x / sigma) * (x / sigma));
    return out;
}

__device__ float2 linear_interp_comp(float const *x, float2 const *y, float const xq, int const y_zero_idx, int const N)
{
    // RC is the whole matrix, has to take only 1 column
    int const y_end_idx = y_zero_idx + N;

    // Manage extrapolation
    if (xq <= x[0])
        return y[0];
    if (xq >= x[N - 1])
        return y[y_end_idx - 1];
    float2 yout;
    // search the left value
    int j = 0;
    while (x[j + 1] < xq)
        j++;
    float xL = x[j], xR = x[j + 1];
    float2 yL = y[y_zero_idx + j], yR = y[y_zero_idx + j + 1];
    float grad_real = (yR.x - yL.x) / (xR - xL);
    float grad_imag = (yR.y - yL.y) / (xR - xL);
    yout.x = yL.x + grad_real * (xq - xL);
    yout.y = yL.y + grad_imag * (xq - xL);
    return yout;
}

__device__ float2 exp_comp(float const ampl, float const phase)
{
    float2 out;
    out.x = ampl * cosf(phase);
    out.y = ampl * sinf(phase);
    return out;
}

__device__ float2 mult_comp(float2 const a, float2 const b)
{
    float2 out;
    out.x = a.x * b.x - a.y * b.y;
    out.y = a.x * b.y + a.y * b.x;
    return out;
}
__global__ void focusTDBPKernel(float const *X, float const *Y, float const z0, float const *TX_pos_x,
                                float const *TX_pos_y, float const *TX_pos_z, float const *RX_pos_x, float const *RX_pos_y, float const *RX_pos_z,
                                float const lambda, float const Dk, float2 const *RC, float const *t, float const f0, float const k_rx_0,
                                float2 *Sn, float *Wn, int const N_pixel, int const N_RC, int const tau, int const squint, float const speed_weight)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= N_pixel)
        return;
    int out_i = i + N_pixel*squint;
    
    float Y_i = Y[i];
    float RX_pos_y_tau = RX_pos_y[tau];
    /*if (Y_i < RX_pos_y_tau)
    {
        // Backlobe of antenna, pixel is 0
        Wn[out_i] += 0.0;
        Sn[out_i].x += 0.0;
        Sn[out_i].y += 0.0;
        return;
    };*/
    float const C = 2.99792458e8;
    float const pi = 3.1415926535897932385;
    float X_i = X[i];
    float TX_pos_x_tau = TX_pos_x[tau];
    float TX_pos_y_tau = TX_pos_y[tau];
    float TX_pos_z_tau = TX_pos_z[tau];
    float RX_pos_x_tau = RX_pos_x[tau];
    float RX_pos_z_tau = RX_pos_z[tau];


    // Compute target wave number
    float R = sqrt((RX_pos_x_tau - X_i) * (RX_pos_x_tau - X_i) + (RX_pos_y_tau - Y_i) * (RX_pos_y_tau - Y_i));
    float psi = asinf((X_i - RX_pos_x_tau) / R);
    float k_rx = sinf(psi) * 2 * pi / lambda;

    // Weight function
    float sigma = Dk / 2;
    
    float Wn_i = speed_weight * gaussActivFunc(k_rx - k_rx_0, sigma);
    // float Wn_i = gaussActivFunc(k_rx - k_rx_0, sigma);
    if (Wn_i < 0.1){
        return;
    }



    // Range distances from the tx antenna [m]
    float R_tx = sqrt((TX_pos_x_tau - X_i) * (TX_pos_x_tau - X_i) + (TX_pos_y_tau - Y_i) * (TX_pos_y_tau - Y_i) + (TX_pos_z_tau - z0) * (TX_pos_z_tau - z0));
    // Range distances from the rx antenna [m]
    float R_rx = sqrt((RX_pos_x_tau - X_i) * (RX_pos_x_tau - X_i) + (RX_pos_y_tau - Y_i) * (RX_pos_y_tau - Y_i) + (RX_pos_z_tau - z0) * (RX_pos_z_tau - z0));
    // Total Tx-target-Rx distance [m]
    float distance = R_tx + R_rx;
    float delay = distance / C;  

    // Backprojection of data from a single Radar position
    int const RC_zero_idx = tau * N_RC;
    float2 RC_1 = linear_interp_comp(t, RC, delay, RC_zero_idx, N_RC);

    float2 RC_2 = mult_comp(RC_1, exp_comp(1, 2 * pi * f0 * delay));
    Sn[out_i].x += Wn_i * RC_2.x;
    Sn[out_i].y += Wn_i * RC_2.y;
    Wn[out_i] += Wn_i;
}

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{
    //========================================= Input variables
    mxGPUArray const *X, *Y, *RC, *t, *TX_pos_x, *TX_pos_y, *TX_pos_z, *RX_pos_x, *RX_pos_y, *RX_pos_z;

    float const *d_X, *d_Y, *d_t, *d_TX_pos_x, *d_TX_pos_y, *d_TX_pos_z, *d_RX_pos_x, *d_RX_pos_y, *d_RX_pos_z,*k_rx_0_vec,*RX_speed;
    float2 const *d_RC;
    //========================================= Constants
    float const *_z0, *_lambda, *_Dk, *_f0,*_median_speed;

    //========================================= Output variables
    mxGPUArray *Sn, *Wn;
    float2 *d_Sn;
    float *d_Wn;
    
    int N_pixel, N_RC, N_tau, N_squints;
    char const *const errId = "parallel:gpu:mexGPUExample:InvalidInput";

    //========================================= Input validation
    if (nrhs != 17 || nlhs != 2)
        mexErrMsgIdAndTxt(errId, "Wrong number of input/output arguments.");
    if (!(mxIsGPUArray(prhs[0]) && mxIsGPUArray(prhs[1])))
        mexErrMsgIdAndTxt(errId, "Input must be GPUArray");

    int const threadsPerBlock = 256;
    int blocksPerGrid;

    mxInitGPU();
    //========================================= Initialize variables
    X = mxGPUCreateFromMxArray(prhs[0]);
    Y = mxGPUCreateFromMxArray(prhs[1]);
    _z0 = (float const *)mxGetData(prhs[2]);
    TX_pos_x = mxGPUCreateFromMxArray(prhs[3]);
    TX_pos_y = mxGPUCreateFromMxArray(prhs[4]);
    TX_pos_z = mxGPUCreateFromMxArray(prhs[5]);
    RX_pos_x = mxGPUCreateFromMxArray(prhs[6]);
    RX_pos_y = mxGPUCreateFromMxArray(prhs[7]);
    RX_pos_z = mxGPUCreateFromMxArray(prhs[8]);
    _lambda = (float const *)mxGetData(prhs[9]);
    _Dk = (float const *)mxGetData(prhs[10]);
    RC = mxGPUCreateFromMxArray(prhs[11]);
    t = mxGPUCreateFromMxArray(prhs[12]);
    _f0 = (float const *)mxGetData(prhs[13]);
    k_rx_0_vec = (float const *) mxGetData(prhs[14]);
    RX_speed = (float const *) mxGetData(prhs[15]);
    _median_speed = (float const *) mxGetData(prhs[16]);

    float const z0 = _z0[0];
    float const lambda = _lambda[0];
    float const Dk = _Dk[0];
    float const f0 = _f0[0];
    float const median_speed = _median_speed[0];

    if (mxGPUGetClassID(X) != mxSINGLE_CLASS)
    {
        mexErrMsgIdAndTxt(errId, "Input must be float");
    }

    //========================================= Initialize pointers
    d_X = (float const *)mxGPUGetDataReadOnly(X);
    d_Y = (float const *)mxGPUGetDataReadOnly(Y);
    d_TX_pos_x = (float const *)mxGPUGetDataReadOnly(TX_pos_x);
    d_TX_pos_y = (float const *)mxGPUGetDataReadOnly(TX_pos_y);
    d_TX_pos_z = (float const *)mxGPUGetDataReadOnly(TX_pos_z);
    d_RX_pos_x = (float const *)mxGPUGetDataReadOnly(RX_pos_x);
    d_RX_pos_y = (float const *)mxGPUGetDataReadOnly(RX_pos_y);
    d_RX_pos_z = (float const *)mxGPUGetDataReadOnly(RX_pos_z);
    d_RC = (float2 const *)mxGPUGetDataReadOnly(RC);
    d_t = (float const *)mxGPUGetDataReadOnly(t);

    //========================================= Create ouput array
    N_pixel = (int)mxGPUGetNumberOfElements(X);
    N_RC = (int)mxGPUGetDimensions(RC)[0];
    N_tau = (int)mxGPUGetDimensions(RC)[1];
    N_squints = (int)mxGetDimensions(prhs[14])[0];
    
    mwSize out_N_dim = 3, out_dims[3] = {mxGPUGetDimensions(X)[0],mxGPUGetDimensions(X)[1],N_squints}; 
    
    Sn = mxGPUCreateGPUArray(out_N_dim,
                             out_dims,
                             mxGPUGetClassID(RC),
                             mxGPUGetComplexity(RC),
                             MX_GPU_INITIALIZE_VALUES);
    d_Sn = (float2 *)mxGPUGetData(Sn);
    Wn = mxGPUCreateGPUArray(out_N_dim,
                             out_dims,
                             mxGPUGetClassID(X),
                             mxGPUGetComplexity(X),
                             MX_GPU_INITIALIZE_VALUES);
    d_Wn = (float *)mxGPUGetData(Wn);

    //========================================= Elaboration

    
    std::cout << "\nN_pix " << N_pixel << ", N_RC " << N_RC << "\n";
    std::cout << "lambda " << lambda << " freq " << f0 << "\n";
    std::cout << "RC dim 1 " << N_RC << "\n";
    std::cout << "RC dim 2 " << N_tau << "\n";
    std::cout << "N squints " << N_squints << "\n";
    
    blocksPerGrid = (N_pixel + threadsPerBlock - 1) / threadsPerBlock;
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());

    float speed_weight;
    for(int squint = 0; squint < N_squints;squint++){
        for (int tau = 0; tau < N_tau; tau++)
        {
            speed_weight = RX_speed[tau] / median_speed;
            focusTDBPKernel<<<blocksPerGrid, threadsPerBlock>>>(
                d_X, d_Y, z0, d_TX_pos_x, d_TX_pos_y, d_TX_pos_z, d_RX_pos_x, d_RX_pos_y, d_RX_pos_z,
                lambda, Dk, d_RC, d_t, f0, k_rx_0_vec[squint], d_Sn, d_Wn, N_pixel, N_RC, tau, squint, speed_weight);
        }
        std::cout << "Squint k_rx " << k_rx_0_vec[squint] << "\n";
        std::cout << "Squint n " << squint +1 << " of " << N_squints << "\n";
    }
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // OUTPUT

    plhs[0] = mxGPUCreateMxArrayOnGPU(Sn);
    plhs[1] = mxGPUCreateMxArrayOnGPU(Wn);

    // Destroy
    mxGPUDestroyGPUArray(Sn);
    mxGPUDestroyGPUArray(Wn);
    mxGPUDestroyGPUArray(X);
    mxGPUDestroyGPUArray(Y);
    mxGPUDestroyGPUArray(TX_pos_x);
    mxGPUDestroyGPUArray(TX_pos_y);
    mxGPUDestroyGPUArray(TX_pos_z);
    mxGPUDestroyGPUArray(RX_pos_x);
    mxGPUDestroyGPUArray(RX_pos_y);
    mxGPUDestroyGPUArray(RX_pos_z);
    mxGPUDestroyGPUArray(RC);
    mxGPUDestroyGPUArray(t);
    //mxGPUDestroyGPUArray(k_rx_0_vec);
}